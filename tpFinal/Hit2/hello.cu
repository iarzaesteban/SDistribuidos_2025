
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU() {
    printf("¡Hola desde el GPU!\n");
}

int main() {
    printf("Hola desde el CPU\n");

    // Llamada al kernel con 1 bloque y 1 hilo
    helloFromGPU<<<1, 1>>>();

    // Esperar a que el GPU termine antes de salir
    hipDeviceSynchronize();

    return 0;
}
