#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/random.h>
#include <hip/hip_runtime.h>

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "Error CUDA en " << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    thrust::default_random_engine rng(1337);
    thrust::uniform_int_distribution<int> dist;
    thrust::host_vector<int> h_vec(32 << 20);
    thrust::generate(h_vec.begin(), h_vec.end(), [&] { return dist(rng); });

    thrust::device_vector<int> d_vec;
    try {
        d_vec = h_vec;
    } catch (thrust::system_error &e) {
        std::cerr << "Error al copiar a dispositivo: " << e.what() << std::endl;
        return EXIT_FAILURE;
    }

    try {
        thrust::sort(d_vec.begin(), d_vec.end());
    } catch (thrust::system_error &e) {
        std::cerr << "Error al ordenar en dispositivo: " << e.what() << std::endl;
        return EXIT_FAILURE;
    }

    thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

    // Check CUDA errors
    checkCudaError(hipDeviceSynchronize(), "sincronización final");

    std::cout << "Primeros 10 valores ordenados:" << std::endl;
    for (int i = 0; i < 10; ++i) {
        std::cout << h_vec[i] << " ";
    }
    std::cout << std::endl;

    return 0;
}
